#include "hip/hip_runtime.h"
/* Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 *
 * This sample demonstrates implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * With compute capability 8.0 or higher the CUDA kernels involved uses asynchronously copy data
 * from global to shared memory; a.k.a., async-copy.
 * This sample has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <cuda_pipeline.h>
#if __CUDA_ARCH__ >= 700
#include <cuda_awbarrier.h>
#endif

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

namespace nvcuda_namespace = nvcuda::experimental;

enum kernels
{
    AsyncCopyMultiStageLargeChunk = 0,
    AsyncCopyLargeChunk           = 1,
    AsyncCopyLargeChunkAWBarrier  = 2,
    AsyncCopyMultiStage           = 3,
    AsyncCopySingleStage          = 4,
    Naive                         = 5,
    NaiveLargeChunk               = 6
};

const char* kernelNames[] = {"AsyncCopyMultiStageLargeChunk", "AsyncCopyLargeChunk", 
                            "AsyncCopyLargeChunkAWBarrier", "AsyncCopyMultiStage", 
                            "AsyncCopySingleStage", "Naive", "NaiveLargeChunk"};

#define USE_CPP_API 0

constexpr int blockSize = 16;

// Multi Stage memcpy_async pipeline with large chunk copy
template <int BLOCK_SIZE> __global__ void MatrixMulAsyncCopyMultiStageLargeChunk(float* __restrict__ C, 
                                                        const float* __restrict__ A,
                                                        const float* __restrict__ B, int wA,
                                                        int wB) {
    // Requires BLOCK_SIZE % 4 == 0 

    // Multi-stage pipeline version
    constexpr size_t maxPipelineStages = 4;

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A for each stage
    __shared__ float As[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B for each stage
    __shared__ float Bs[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

    float Csub = 0.0;

    // Index of the first sub-matrix of A processed by the block
    const int aBegin = wA * (BLOCK_SIZE) * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    const int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    const int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    const int t4x = threadIdx.x * 4 ;

#if USE_CPP_API
    nvcuda_namespace::pipeline pipe;
#endif
    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin, i = 0, aStage = aBegin, bStage = bBegin, iStage = 0; a <= aEnd; a += aStep, b += bStep, ++i ) {
        // Load the matrices from device memory to shared memory; each thread loads
        // one element of each matrix

        for ( ; aStage <= a + aStep * maxPipelineStages ; aStage += aStep, bStage += bStep, ++iStage )
        {
            if ( aStage <= aEnd && t4x < BLOCK_SIZE )
            {
                // Rotating buffer
                const int j = iStage % maxPipelineStages;
                float4 * const A4s = reinterpret_cast<float4*>(& As[j][threadIdx.y][t4x]);
                float4 * const B4s = reinterpret_cast<float4*>(& Bs[j][threadIdx.y][t4x]);
                const float4 * const A4  = reinterpret_cast<const float4*>(& A[aStage + wA * threadIdx.y + t4x]);
                const float4 * const B4  = reinterpret_cast<const float4*>(& B[aStage + wA * threadIdx.y + t4x]);

#if USE_CPP_API
                nvcuda_namespace::memcpy_async(*A4s,*A4, pipe);
                nvcuda_namespace::memcpy_async(*B4s,*B4, pipe);
#else
                __pipeline_memcpy_async(A4s, A4, sizeof(float4));
                __pipeline_memcpy_async(B4s, B4, sizeof(float4));
#endif
            }

#if USE_CPP_API
            pipe.commit();
#else
            __pipeline_commit();
#endif
        }
#if USE_CPP_API
        pipe.wait_prior<maxPipelineStages-1>();
#else
        __pipeline_wait_prior(maxPipelineStages-1);
#endif
        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Rotating buffer
        const int j = i % maxPipelineStages;

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[j][threadIdx.y][k] * Bs[j][k][threadIdx.x];
        }

        // Don't have to synchronize because 
        // next iteration is loading to a different buffer
    }

    // Write the block sub-matrix to device memory;
    // each thread writes four element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Single Stage memcpy_async pipeline with Large copy chunk (float4)
template <int BLOCK_SIZE> __global__ void MatrixMulAsyncCopyLargeChunk(float* __restrict__ C, 
                                                        const float* __restrict__ A,
                                                        const float* __restrict__ B, int wA,
                                                        int wB) {
    // Requires BLOCK_SIZE % 4 == 0 

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Single-stage pipeline version
    float Csub = 0.0;

    const int t4x = threadIdx.x * 4;
#if USE_CPP_API
    nvcuda_namespace::pipeline pipe;
#endif
    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Load the matrices from device memory to shared memory; 
        // a subset of threads loads a contiguous chunk of elements.

        // Previously, per-thread:
        // As[ty][tx] = A[a + wA * ty + tx];
        // Bs[ty][tx] = B[b + wB * ty + tx];

        // Now, one fourth of the threads load four elements of each matrix
        if ( t4x < BLOCK_SIZE ) {
            float4 * const A4s = reinterpret_cast<float4*>(& As[threadIdx.y][t4x]);
            float4 * const B4s = reinterpret_cast<float4*>(& Bs[threadIdx.y][t4x]);
            const float4 * const A4  = reinterpret_cast<const float4*>(& A[a + wA * threadIdx.y + t4x]);
            const float4 * const B4  = reinterpret_cast<const float4*>(& B[a + wA * threadIdx.y + t4x]);

#if USE_CPP_API
            nvcuda_namespace::memcpy_async(*A4s,*A4,pipe);
            nvcuda_namespace::memcpy_async(*B4s,*B4,pipe);

            pipe.commit_and_wait();
#else
            __pipeline_memcpy_async(A4s, A4, sizeof(float4));
            __pipeline_memcpy_async(B4s, B4, sizeof(float4));

            __pipeline_commit();
            __pipeline_wait_prior(0);
#endif
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes four element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Single Stage memcpy_async pipeline with Large copy chunk (float4) using arrive-wait barrier
template <int BLOCK_SIZE> __global__ void MatrixMulAsyncCopyLargeChunkAWBarrier(float* __restrict__ C, 
                                                        const float* __restrict__ A,
                                                        const float* __restrict__ B, int wA,
                                                        int wB) {
#if __CUDA_ARCH__ >= 700
    // Requires BLOCK_SIZE % 4 == 0 

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    nvcuda_namespace::pipeline pipe;
    __shared__ nvcuda_namespace::awbarrier barrier;

    if (threadIdx.x == 0) {
        nvcuda_namespace::init(&barrier, blockDim.x*blockDim.y);
    }
    __syncthreads();

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    float Csub = 0.0;

    const int t4x = threadIdx.x * 4;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Load the matrices from device memory to shared memory; 
        // a subset of threads loads a contiguous chunk of elements.

        // Now, one fourth of the threads load four elements of each matrix
        if ( t4x < BLOCK_SIZE ) {
            float4 * const A4s = reinterpret_cast<float4*>(& As[threadIdx.y][t4x]);
            float4 * const B4s = reinterpret_cast<float4*>(& Bs[threadIdx.y][t4x]);
            const float4 * const A4  = reinterpret_cast<const float4*>(& A[a + wA * threadIdx.y + t4x]);
            const float4 * const B4  = reinterpret_cast<const float4*>(& B[a + wA * threadIdx.y + t4x]);

            nvcuda_namespace::memcpy_async(*A4s,*A4,pipe);
            nvcuda_namespace::memcpy_async(*B4s,*B4,pipe);

            pipe.arrive_on(barrier);
        }

        // Synchronize to make sure the matrices are loaded
        barrier.arrive_and_wait();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes four element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
#endif
}

// Single Stage memcpy_async pipeline with float copy
template <int BLOCK_SIZE> __global__ void MatrixMulAsyncCopySingleStage(float *C, const float *A,
                                                        const float *B, int wA,
                                                        int wB) {

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Single-stage pipeline version
    float Csub = 0.0;

#if USE_CPP_API
    nvcuda_namespace::pipeline pipe;
#endif
    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Load the matrices from device memory to shared memory; each thread loads
        // one element of each matrix
        {
            const float *A_float = reinterpret_cast<const float*>(A + a + wA * threadIdx.y + threadIdx.x);
            const float *B_float = reinterpret_cast<const float*>(B + b + wB * threadIdx.y + threadIdx.x);

#if USE_CPP_API

            nvcuda_namespace::memcpy_async(As[threadIdx.y][threadIdx.x], *A_float, pipe);
            nvcuda_namespace::memcpy_async(Bs[threadIdx.y][threadIdx.x], *B_float, pipe);

            pipe.commit_and_wait();
#else
            __pipeline_memcpy_async(&As[threadIdx.y][threadIdx.x], A_float, sizeof(float));
            __pipeline_memcpy_async(&Bs[threadIdx.y][threadIdx.x], B_float, sizeof(float));

            __pipeline_commit();
            __pipeline_wait_prior(0);
#endif
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes four element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Multi Stage memcpy_async pipeline with int copy
template <int BLOCK_SIZE> __global__ void MatrixMulAsyncCopyMultiStage(float* __restrict__ C, 
                                                        const float* __restrict__ A,
                                                        const float* __restrict__ B, int wA,
                                                        int wB) {
    // Multi-stage pipeline version
    constexpr size_t maxPipelineStages = 4;

    // Declaration of the shared memory array As used to
    // store the sub-matrix of A for each stage
    __shared__ float As[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B for each stage
    __shared__ float Bs[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

    float Csub = 0.0;

    // Index of the first sub-matrix of A processed by the block
    const int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    const int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    const int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

#if USE_CPP_API
    nvcuda_namespace::pipeline pipe;
#endif
    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin, i = 0, aStage = aBegin, bStage = bBegin, iStage = 0; a <= aEnd; a += aStep, b += bStep, ++i ) {
        // Load the matrices from device memory to shared memory; each thread loads
        // one element of each matrix

        for ( ; aStage <= a + aStep * maxPipelineStages ; aStage += aStep, bStage += bStep, ++iStage )
        {
            if ( aStage <= aEnd )
            {
                const float *A_float = reinterpret_cast<const float*>(A + aStage + wA * threadIdx.y + threadIdx.x);
                const float *B_float = reinterpret_cast<const float*>(B + bStage + wB * threadIdx.y + threadIdx.x);

                // Rotating buffer
                const int j = iStage % maxPipelineStages;
#if USE_CPP_API
                nvcuda_namespace::memcpy_async(As[j][threadIdx.y][threadIdx.x], *A_float, pipe);
                nvcuda_namespace::memcpy_async(Bs[j][threadIdx.y][threadIdx.x], *B_float, pipe);
#else
                __pipeline_memcpy_async(&As[j][threadIdx.y][threadIdx.x], A_float, sizeof(float));
                __pipeline_memcpy_async(&Bs[j][threadIdx.y][threadIdx.x], B_float, sizeof(float));
#endif
            }
#if USE_CPP_API
            pipe.commit();
#else
            __pipeline_commit();
#endif
        }
#if USE_CPP_API
        pipe.wait_prior<maxPipelineStages-1>();
#else
        __pipeline_wait_prior(maxPipelineStages-1);
#endif
        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        const int j = i % maxPipelineStages;

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[j][threadIdx.y][k] * Bs[j][k][threadIdx.x];
        }

        // Don't have to synchronize because 
        // next iteration is loading to a different buffer
    }

    // Write the block sub-matrix to device memory;
    // each thread writes four element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulNaive(float *C, float *A,
                                                        float *B, int wA,
                                                        int wB) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[threadIdx.y][threadIdx.x] = A[a + wA * threadIdx.y + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[b + wB * threadIdx.y + threadIdx.x];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

template <int BLOCK_SIZE> __global__ void MatrixMulNaiveLargeChunk(float *C, float *A,
                                                        float *B, int wA,
                                                        int wB) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int t4x = threadIdx.x * 4 ;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * blockIdx.y;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * blockIdx.x;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {

        // Load the matrices from device memory
        // to shared memory; 

        // One fourth of the threads load four elements of each matrix
        if ( t4x < BLOCK_SIZE ) {
            float4 * const A4s = reinterpret_cast<float4*>(& As[threadIdx.y][t4x]);
            float4 * const B4s = reinterpret_cast<float4*>(& Bs[threadIdx.y][t4x]);
            const float4 * const A4 = reinterpret_cast<float4*>(& A[a + wA * threadIdx.y + t4x]);
            const float4 * const B4 = reinterpret_cast<float4*>(& B[a + wA * threadIdx.y + t4x]);
            *A4s = *A4 ;
            *B4s = *B4 ;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}


void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

/**
 * Run matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   const dim3 &dimsA,
                   const dim3 &dimsB,
                   kernels kernel_number) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));
    hipStream_t stream;

    // Initialize host memory
    const float valB = 2.10f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemsetAsync(d_C, 0, mem_size_C, stream));

    // Setup execution parameters
    dim3 threads(blockSize, blockSize);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);


    printf("Running kernel = %d - %s\n", kernel_number, kernelNames[kernel_number]);
    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    switch (kernel_number)
    {
        case AsyncCopyMultiStageLargeChunk :
        default:
            MatrixMulAsyncCopyMultiStageLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case AsyncCopyLargeChunk :
            MatrixMulAsyncCopyLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case AsyncCopyLargeChunkAWBarrier :
            MatrixMulAsyncCopyLargeChunkAWBarrier<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case AsyncCopyMultiStage :
            MatrixMulAsyncCopyMultiStage<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case AsyncCopySingleStage :
            MatrixMulAsyncCopySingleStage<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case Naive :
            MatrixMulNaive<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
        case NaiveLargeChunk:
            MatrixMulNaiveLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
            break;
    }

    printf("done\n");
    checkCudaErrors(hipStreamSynchronize(stream));


    // Execute the kernel
    int nIter = 100;

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream));

    for (int j = 0; j < nIter; j++) {
        switch (kernel_number)
        {
            case AsyncCopyMultiStageLargeChunk :
            default:
                MatrixMulAsyncCopyMultiStageLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case AsyncCopyLargeChunk :
                MatrixMulAsyncCopyLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case AsyncCopyLargeChunkAWBarrier :
                MatrixMulAsyncCopyLargeChunkAWBarrier<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case AsyncCopyMultiStage :
                MatrixMulAsyncCopyMultiStage<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case AsyncCopySingleStage :
                MatrixMulAsyncCopySingleStage<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case Naive :
                MatrixMulNaive<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
            case NaiveLargeChunk:
                MatrixMulNaiveLargeChunk<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
                break;
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                               static_cast<double>(dimsA.y) *
                               static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                       (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    // |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    printf("\nNOTE: The CUDA Samples are not meant for performance"\
           "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}


int main(int argc, char **argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("      -kernel=kernel_number (0 - AsyncCopyMultiStageLargeChunk; 1 - AsyncCopyLargeChunk)\n");
        printf("                            (2 - AsyncCopyLargeChunkAWBarrier; 3 - AsyncCopyMultiStage)\n");
        printf("                            (4 - AsyncCopySingleStage; 5 - Naive without memcpy_async)\n");
        printf("                            (6 - NaiveLargeChunk without memcpy_async)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int matrixBlock = 32;
    dim3 dimsA(10 * 2 * matrixBlock, 10 * 2 * matrixBlock, 1);
    dim3 dimsB(10 * 2 * matrixBlock, 10 * 2 * matrixBlock, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    kernels selected_kernel = AsyncCopyMultiStageLargeChunk;

    // kernel to run - default (AsyncCopyMultiStageLargeChunk == 0)
    if (checkCmdLineFlag(argc, (const char **)argv, "kernel")) {
        int kernel_number = getCmdLineArgumentInt(argc, (const char **)argv, "kernel");
        if (kernel_number < 7)
        {
            selected_kernel = (kernels)kernel_number;
        }
        else
        {
            printf("Error: kernel number should be between 0 to 6, you have entered %d\n", kernel_number);
            exit(EXIT_FAILURE);
        }

        int major = 0;
        checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, dev));
        if ((kernel_number == AsyncCopyLargeChunkAWBarrier) && major < 7)
        {
            printf("AsyncCopyLargeChunkAWBarrier kernel requires requires SM 7.0 or higher.  Exiting...\n");
            exit(EXIT_WAIVED);
        }
    }


    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
                                               dimsB.x, dimsB.y);

    int matrix_result = MatrixMultiply(argc, argv, dimsA, dimsB, selected_kernel);

    exit(matrix_result);
}

